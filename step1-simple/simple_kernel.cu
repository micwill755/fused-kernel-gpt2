// Step 1: Simple CUDA Kernel - Element-wise Addition
// Compile directly with: nvcc -o simple_kernel simple_kernel.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

// CUDA kernel function - runs on GPU
// A kernel is simply parallel code that runs on the GPU
__global__ void simple_add_kernel(
    const float* input_a,    // Input tensor A - in this example tensors are only a 1d array, but in future
                             // examples these will become multi-dimensional tensors - GPT-2 operations: attention matrices, embeddings, layer outputs
    const float* input_b,    // Input tensor B  
    float* output,           // Output tensor
    int num_elements         // Total number of elements
) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Boundary check - make sure we don't go out of bounds
    if (idx < num_elements) {
        // Simple element-wise addition
        output[idx] = input_a[idx] + input_b[idx];
    }
}

// Helper function to check CUDA errors
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)

// Initialize array with random values
void init_array(float* arr, int size) {
    for (int i = 0; i < size; i++) {
        arr[i] = (float)rand() / RAND_MAX * 2.0f - 1.0f; // Random values between -1 and 1
    }
}

// Verify results by comparing with CPU computation
bool verify_results(const float* a, const float* b, const float* gpu_result, int size) {
    const float tolerance = 1e-5f;
    
    for (int i = 0; i < size; i++) {
        float expected = a[i] + b[i];
        float diff = fabsf(gpu_result[i] - expected);
        
        if (diff > tolerance) {
            printf("Verification failed at index %d: expected %f, got %f (diff: %f)\n", 
                   i, expected, gpu_result[i], diff);
            return false;
        }
    }
    return true;
}

// Benchmark function
double benchmark_kernel(float* d_a, float* d_b, float* d_output, int size, int iterations) {
    const int threads_per_block = 256;
    const int blocks = (size + threads_per_block - 1) / threads_per_block;
    
    // Warm up - we create 10 identical runs of the same simple_add_kernel that will be timed later.
    for (int i = 0; i < 10; i++) {
        simple_add_kernel<<<blocks, threads_per_block>>>(d_a, d_b, d_output, size);
    }
    // cudaDeviceSynchronize() forces the CPU to wait for all GPU work to complete
    CUDA_CHECK(hipDeviceSynchronize()); 
    
    // Timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start));
    for (int i = 0; i < iterations; i++) {
        simple_add_kernel<<<blocks, threads_per_block>>>(d_a, d_b, d_output, size);
    }
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    
    return milliseconds / iterations; // Average time per iteration
}

int main() {
    printf("Simple CUDA Kernel - Element-wise Addition\n");
    printf("==============================================\n");
    
    // Initialize CUDA
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    
    if (device_count == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    printf("GPU: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
    
    // Test different array sizes
    int sizes[] = {10000, 1000000, 25000000}; // 10K, 1M, 25M elements
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);
    
    srand(time(NULL)); // Initialize random seed
    
    for (int s = 0; s < num_sizes; s++) {
        int size = sizes[s];
        size_t bytes = size * sizeof(float);
        
        printf("\n Testing with %d elements (%.2f MB)\n", size, bytes / (1024.0f * 1024.0f));
        
        // Allocate host memory
        float* h_a = (float*)malloc(bytes);
        float* h_b = (float*)malloc(bytes);
        float* h_output = (float*)malloc(bytes);
        
        if (!h_a || !h_b || !h_output) {
            printf("Failed to allocate host memory\n");
            return 1;
        }
        
        // Initialize input arrays
        init_array(h_a, size);
        init_array(h_b, size);
        
        // Allocate device memory
        float* d_a;
        float* d_b;
        float* d_output;
        
        CUDA_CHECK(hipMalloc(&d_a, bytes));
        CUDA_CHECK(hipMalloc(&d_b, bytes));
        CUDA_CHECK(hipMalloc(&d_output, bytes));
        
        // Copy data to device
        CUDA_CHECK(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));
        
        // Launch kernel
        const int threads_per_block = 256;
        const int blocks = (size + threads_per_block - 1) / threads_per_block;
        
        printf("Launching kernel: %d blocks × %d threads = %d total threads\n", 
               blocks, threads_per_block, blocks * threads_per_block);
        
        // The <<<>>> syntax launches the kernel on the GPU - syntax can take up to 4 parameters, but only the first 2 are required:
        // e.g. kernel <<<grid_size, block_size, shared_mem, stream>>>(args);
        // grid_size (required): Number of thread blocks
        // block_size (required): Threads per block
        // shared_mem (optional): Shared memory bytes per block (default: 0)
        // stream (optional): CUDA stream for async execution (default: 0)
        simple_add_kernel<<<blocks, threads_per_block>>>(d_a, d_b, d_output, size);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Copy result back to host
        CUDA_CHECK(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
        
        // Verify results
        if (verify_results(h_a, h_b, h_output, size)) {
            printf("Correctness test PASSED!\n");
        } else {
            printf("Correctness test FAILED!\n");
            return 1;
        }
        
        // Benchmark performance
        double avg_time = benchmark_kernel(d_a, d_b, d_output, size, 100);
        double bandwidth = (3.0 * bytes) / (avg_time * 1e-3) / 1e9; // GB/s (3 arrays: 2 reads + 1 write)
        
        printf("⚡ Performance: %.3f ms (%.2f GB/s)\n", avg_time, bandwidth);
        
        // Cleanup
        free(h_a);
        free(h_b);
        free(h_output);
        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_b));
        CUDA_CHECK(hipFree(d_output));
    }
    
    printf("\nAll tests completed successfully!\n");

    return 0;
}
